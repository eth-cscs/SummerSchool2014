
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdint.h>

#define CUDA_ERR_CHECK(x) \
        do { hipError_t err = x; if (err != hipSuccess) { \
                fprintf (stderr, "Error \"%s\" at %s:%d \n", \
                 hipGetErrorString(err), \
                __FILE__, __LINE__); exit(-1); \
        }} while (0);

#define BLOCK_SIZE 256

struct SoA
{
    uint8_t *r;
    uint8_t *g;
    uint8_t *b;
};

__global__ void kernel_SOA(uint8_t* r, uint8_t* g, uint8_t* b, int N)
{
	int idxx = threadIdx.x + blockIdx.x * blockDim.x;
	
	if (idxx < N)
	{
		uint8_t b1, b2, b3, res;
		b1 = r[idxx];
		b2 = g[idxx];
		b3 = b[idxx];
		res = 0.2126f * b1 + 0.7152f * b2 + 0.0722f * b3;
		r[idxx] = res;
		g[idxx] = res;
		b[idxx] = res;
	}
}

int main()
{
	using namespace std;

	int N;
	cout << "input N" << endl;
	cin >> N;
	
	hipEvent_t start, stop;
	CUDA_ERR_CHECK(hipEventCreate(&start));
	CUDA_ERR_CHECK(hipEventCreate(&stop));
	
	float time;
	size_t size;
	dim3 block;
	block.x = BLOCK_SIZE;
	dim3 grid(((N - 1) / block.x + 1), 1, 1);

	size = N * sizeof(uint8_t);

	SoA* hSoA;
	hSoA = new SoA();
	hSoA->r = (uint8_t*)malloc(size);
	hSoA->g = (uint8_t*)malloc(size);
	hSoA->b = (uint8_t*)malloc(size);
	
	uint8_t *r, *g, *b;
	CUDA_ERR_CHECK(hipMalloc((void**)&r,size));
	CUDA_ERR_CHECK(hipMalloc((void**)&g,size));
	CUDA_ERR_CHECK(hipMalloc((void**)&b,size));
	
	CUDA_ERR_CHECK(hipMemcpy(r,hSoA->r,size, hipMemcpyHostToDevice));
	CUDA_ERR_CHECK(hipMemcpy(g,hSoA->g,size, hipMemcpyHostToDevice));
	CUDA_ERR_CHECK(hipMemcpy(b,hSoA->b,size, hipMemcpyHostToDevice));
	
	CUDA_ERR_CHECK(hipEventRecord(start));
	
	kernel_SOA<<<grid,block>>>(r, g, b, N);
	CUDA_ERR_CHECK(hipGetLastError());
	
	CUDA_ERR_CHECK(hipEventRecord(stop));
	CUDA_ERR_CHECK(hipDeviceSynchronize());
	CUDA_ERR_CHECK(hipEventElapsedTime(&time, start, stop));
	
	CUDA_ERR_CHECK(hipMemcpy(hSoA->r,r,size, hipMemcpyDeviceToHost));
	CUDA_ERR_CHECK(hipMemcpy(hSoA->g,g,size, hipMemcpyDeviceToHost));
	CUDA_ERR_CHECK(hipMemcpy(hSoA->b,b,size, hipMemcpyDeviceToHost));
	
	cout << endl << time << endl;
	
	CUDA_ERR_CHECK(hipFree(r));
	CUDA_ERR_CHECK(hipFree(g));
	CUDA_ERR_CHECK(hipFree(b));

	CUDA_ERR_CHECK(hipEventDestroy(start));
	CUDA_ERR_CHECK(hipEventDestroy(stop));

	free(hSoA);
	
	return 0;
}

